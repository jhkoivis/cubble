#include "Util.h"
#include <hip/hip_runtime.h>
#include <exception>
#include <iostream>
#include <stdexcept>
#include <string>

#include "tests.cuh"
#include "Kernels.cuh"

using namespace std;
using namespace hipcubble;

namespace ASSERT{
  int EQUAL(int x, int y){
	  if (x == y){
	  	cout << "Passed" << endl;
		  return 1;
	  }
	  else{
	  	cout << "Not passed" << endl;
		  return 0;
	  }
  }
};

namespace TestSuite{
using namespace ASSERT;

void TEST_tests_are_running (void)
{
	cout << "Tests ready to start" << endl;
}

void TEST_cubble_simple (void)
{
	cout << test_me() << endl;
	cout << EQUAL(1, 1) << endl;
}

void TEST_simple_cuda (void)
{
	int N = 128*16;
	int *x, *y, *z;
	int *x_d, *y_d, *z_d;

	x = (int *)malloc(sizeof(int) * N);
	y = (int *)malloc(sizeof(int) * N);
	z = (int *)malloc(sizeof(int) * N);
	hipMalloc( (void **)&x_d, sizeof(int) * N);
	hipMalloc( (void **)&y_d, sizeof(int) * N);
	hipMalloc( (void **)&z_d , sizeof(int) * N);

	// initialize x and y arrays on the host
	for (int i = 0; i < N; i++) {
		x[i] = 1+i;
		y[i] = 2;
	}

	hipMemcpy(x_d, x, sizeof(int) * N, hipMemcpyHostToDevice);
	hipMemcpy(y_d, y, sizeof(int) * N, hipMemcpyHostToDevice);

	// Run kernel on 1M elements on the GPU
	cuda_test<<<16, 128>>>(x_d, y_d, z_d);
	hipDeviceSynchronize();

	hipMemcpy(z, z_d, sizeof(int) * N, hipMemcpyDeviceToHost);

	for(int i = 0; i < N; i++){
		cout << "z = " << z[i] << " for i = " << i << endl;
	}

	// Free memory

	hipFree(x_d);
	hipFree(y_d);
	hipFree(z_d);

	free(x);
	free(y);
	free(z);
}

};

/*
__device__ void eulerIntegrate(int idx, double timeStep, double *y, double *f)
{
	y[idx] += f[idx] * timeStep;
}*/
