#include "hip/hip_runtime.h"
#include "Util.h"
#include <hip/hip_runtime.h>
#include <math.h>

#include "cubble_tests.cuh"
#include "Kernels.cuh"
#include "gtest/gtest.h"


__global__ void test_vec_add_cubble(float* x, float* y, float* z, int N){

	cubble::test_vec_add(x, y, z, N);

}


TEST(TestGtestWorkingSuite, AddArrays){

	int N = 1e6;
	float *x, *y, *z;
	float *x_d, *y_d, *z_d;
	double error_tolerance = 1e-5;

	x = (float *)malloc(sizeof(float) * N);
	y = (float *)malloc(sizeof(float) * N);
	z = (float *)malloc(sizeof(float) * N);

	hipMalloc( (void **)&x_d, sizeof(float) * N);
	hipMalloc( (void **)&y_d, sizeof(float) * N);
	hipMalloc( (void **)&z_d , sizeof(float) * N);

	// initialize x and y arrays
	for (int i = 0; i < N; i++) {
		x[i] = sin(i);
		y[i] = cos(i);
	}

	hipMemcpy(x_d, x, sizeof(float) * N, hipMemcpyHostToDevice);
	hipMemcpy(y_d, y, sizeof(float) * N, hipMemcpyHostToDevice);

	dim3 gridDim((int)ceil(N / 256.0));
	dim3 blockDim(256);

	test_vec_add_cubble<<<gridDim, blockDim>>>(x_d, y_d, z_d, N);

	hipDeviceSynchronize();

	hipMemcpy(z, z_d, sizeof(float) * N, hipMemcpyDeviceToHost);

	for(int i = 0; i < N; i++){
		EXPECT_NEAR(x[i] + y[i], z[i], error_tolerance);
	}

	// Free memory

	hipFree(x_d);
	hipFree(y_d);
	hipFree(z_d);

	free(x);
	free(y);
	free(z);

}
